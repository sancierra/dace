#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>



DACE_EXPORTED int __dace_init_cuda(float * __restrict__ A, float * __restrict__ C, int N);
DACE_EXPORTED void __dace_exit_cuda(float * __restrict__ A, float * __restrict__ C, int N);

DACE_DFI void nested_stencil2d_transient_copyin_1_1_1_2_0(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

DACE_DFI void nested_stencil2d_transient_copyin_1_2_1_3_0(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

DACE_DFI void nested_stencil2d_transient_copyin_1_1_4_0(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (2 * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

DACE_DFI void nested_stencil2d_transient_copyin_0_0_8(const float * __in_gpu_A, float* __out_B, int N, int stencil_i, int stencil_j) {

    {
        
        
        nested_stencil2d_transient_copyin_1_1_1_2_0(&__in_gpu_A[(N + 1)], &__in_gpu_A[N], &__in_gpu_A[N], &__in_gpu_A[N], &__in_gpu_A[0], &__out_B[0], N, stencil_j);
    }
    {
        
        
        nested_stencil2d_transient_copyin_1_2_1_3_0(&__in_gpu_A[((2 * N) + 1)], &__in_gpu_A[(2 * N)], &__in_gpu_A[(2 * N)], &__in_gpu_A[(2 * N)], &__in_gpu_A[N], &__out_B[3], N, stencil_j);
    }
    {
        
        
        nested_stencil2d_transient_copyin_1_1_4_0(&__in_gpu_A[((3 * N) + 1)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(2 * N)], &__out_B[6], N, stencil_j);
    }
    
}



namespace dace { namespace cuda {
    hipStream_t __streams[1];
    hipEvent_t __events[2];
    int num_streams = 1;
    int num_events = 2;
} }

int __dace_init_cuda(float * __restrict__ A, float * __restrict__ C, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    hipMalloc((void **) &dev_X, 1);

    // Create cuda streams and events
    for(int i = 0; i < 1; ++i) {
        hipStreamCreateWithFlags(&dace::cuda::__streams[i], hipStreamNonBlocking);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventCreateWithFlags(&dace::cuda::__events[i], hipEventDisableTiming);
    }

    

    return 0;
}

void __dace_exit_cuda(float * __restrict__ A, float * __restrict__ C, int N) {
    

    // Destroy cuda streams and events
    for(int i = 0; i < 1; ++i) {
        hipStreamDestroy(dace::cuda::__streams[i]);
    }
    for(int i = 0; i < 2; ++i) {
        hipEventDestroy(dace::cuda::__events[i]);
    }
}

__global__ void outer_fused_0_0_9(const float * __restrict__ gpu_A, float * __restrict__ gpu_C, int N) {
    {
        {
            int stencil_j = (blockIdx.x * 32 + threadIdx.x);
            int stencil_i = (blockIdx.y * 1 + threadIdx.y);
            float *B = new float DACE_ALIGN(64)[9];
            memset(B, 0, sizeof(float)*9);
            if (stencil_j < (N - 4)) {
                {
                    nested_stencil2d_transient_copyin_0_0_8(&gpu_A[((N * stencil_i) + stencil_j)], &B[0], N, stencil_i, stencil_j);
                    {
                        for (auto i = (stencil_i + 2); i < (stencil_i + 3); i += 1) {
                            for (auto j = (stencil_j + 2); j < (stencil_j + 3); j += 1) {
                                {
                                    float a1 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 4)];
                                    float a2 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 5)];
                                    float a3 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 3)];
                                    float a4 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 1)];
                                    float a5 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 7)];
                                    float b;

                                    ///////////////////
                                    // Tasklet code (b)
                                    b = (2 * ((((a1 + a2) + a3) + a4) + a5));
                                    ///////////////////

                                    gpu_C[((N * i) + j)] = b;
                                }
                            }
                        }
                    }
                    delete[] B;
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_outer_fused_0_0_9(const float * __restrict__ gpu_A, float * __restrict__ gpu_C, int N);
void __dace_runkernel_outer_fused_0_0_9(const float * __restrict__ gpu_A, float * __restrict__ gpu_C, int N)
{

    void  *outer_fused_0_0_9_args[] = { (void *)&gpu_A, (void *)&gpu_C, (void *)&N };
    hipLaunchKernel((void*)outer_fused_0_0_9, dim3(int_ceil(int_ceil((N - 4), 1), 32), int_ceil(int_ceil((N - 4), 1), 1), int_ceil(1, 1)), dim3(32, 1, 1), outer_fused_0_0_9_args, 0, dace::cuda::__streams[0]);
}

