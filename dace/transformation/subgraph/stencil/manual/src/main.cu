/* DaCe AUTO-GENERATED FILE. DO NOT MODIFY */
#include <iostream>
#include <random>
#include <chrono>

#include "kernels.h"

int N = 512;
typedef double dtype;

void run(){
    std::cout << "---Runner---" << std::endl;
    std::cout << "Allocating Arrays...." << std::endl;
    int N = 512;
    dtype* A = new dtype[N][N];
    for(int i = 0; i<N; ++i){
        for(int j=0; j<N; ++j){
            A[i][j] = dtype(i*j)/N
        }
    }

    double* gpu_A = 0;
    double* gpu_B = 0;
    double* gpu_C = 0;

    hipMalloc(&gpu_A, (N*N)*sizeof(dtype));
    hipMalloc(&gpu_B, (N*N)*sizeof(dtype));
    hipMalloc(&gpu_C, (N*N)*sizeof(dtype));

    // run the fused version
    std::cout << "Running Unfused Kernels" << std::endl;
    hipStream_t* stream;
    hipStreamCreate(stream);

    hipMemcpyAsync(gpu_A, A, (N * N) * sizeof(dtype), hipMemcpyHostToDevice, stream0);
    auto start = high_resolution_clock::now();
    run_kernel1(gpu_A, gpu_B, N);
    run_kernel2(gpu_B, gpu_C, N);
    auto end = high_resolution_clock::now();
    std::cout << "Timer: " << duration_cast<microseconds>(end-start).count() << std::endl;

    hipMemcpyAsync(C, gpu_C, (N * N) * sizeof(dtype), hipMemcpyDeviceToHost, stream0);
    hipStreamSynchronize(stream0);

    // run the non-fused version
    std::cout << "Running Fused Kernels" << std::endl;
    hipMemcpyAsync(gpu_A, A, (N * N) * sizeof(dtype), hipMemcpyHostToDevice, stream0);
    start = high_resolution_clock::now();
    run_kernel_fused(gpu_A, gpu_C);
    end = high_resolution_clock::now();
    std::cout << "Timer: " << duration_cast<microseconds>(end-start).count() << std::endl;

    hipMemcpyAsync(C, gpu_C, (N * N) * sizeof(dtype), hipMemcpyDeviceToHost, stream0);
    hipStreamSynchronize(stream0);

    hipFree(gpu_A);
    hipFree(gpu_C);
    std::cout << "Done."  << std::endl;

}

int main(){
    run();
}
