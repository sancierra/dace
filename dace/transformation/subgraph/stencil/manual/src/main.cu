#include "hip/hip_runtime.h"
/* DaCe AUTO-GENERATED FILE. DO NOT MODIFY */
#include <iostream>
#include <random>
#include <chrono>
#include <cmath>
#include <algorithm>

#include "kernels.h"

const int N = 512;
const int TREPS = 1;


void run(){
    std::cout << "---Runner---" << std::endl;
    std::cout << "Allocating Arrays...." << std::endl;

    dtype* A = new dtype[N*N];
    dtype* C = new dtype[N*N];

    dtype* result1 = new dtype[N*N];
    dtype* result2 = new dtype[N*N];

    for(int i = 0; i<N; ++i){
        for(int j=0; j<N; ++j){
            A[i*N + j] = dtype(i*j)/(N*N);
        }
    }

    dtype* gpu_A = 0;
    dtype* gpu_B = 0;
    dtype* gpu_C1 = 0;
    dtype* gpu_C2 = 0;

    hipMalloc(&gpu_A, (N*N)*sizeof(dtype));
    hipMalloc(&gpu_B, (N*N)*sizeof(dtype));
    hipMalloc(&gpu_C1, (N*N)*sizeof(dtype));
    hipMalloc(&gpu_C2, (N*N)*sizeof(dtype));

    // run the non fused version
    std::cout << "Running Unfused Kernels" << std::endl;
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(gpu_A, A, (N * N) * sizeof(dtype), hipMemcpyHostToDevice, stream);
    hipMemset(gpu_C1, 0, N*N*sizeof(dtype));
    // #########################
    auto runtimes = std::vector<double>(TREPS);
    for(int run=0; run<TREPS; run++){
        auto start = std::chrono::high_resolution_clock::now();
        run_kernel1(gpu_A, gpu_B, N, stream);
        run_kernel2(gpu_B, gpu_C1, N, stream);
        auto end = std::chrono::high_resolution_clock::now();
        runtimes[run] = (std::chrono::duration_cast<std::chrono::microseconds>(end-start).count());

    }
    // #########################
    std::sort(runtimes.begin(), runtimes.end());
    
    std::fill(C, C+N*N, 0);
    hipMemcpyAsync(C, gpu_C1, (N * N) * sizeof(dtype), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    std::copy(C, C+N*N, result1);
    
    
    // run the non-fused version
    std::cout << "Running Fused Kernels" << std::endl;
    hipMemcpyAsync(gpu_A, A, (N * N) * sizeof(dtype), hipMemcpyHostToDevice, stream);
    hipMemset(gpu_C2, 0, N*N*sizeof(dtype));
    // ##########################
    runtimes.empty();
    for(int run=0; run<TREPS; run++){
        auto start = std::chrono::high_resolution_clock::now();
        run_fused(gpu_A, gpu_C2, N, stream);
        auto end = std::chrono::high_resolution_clock::now();
        runtimes[run] = (std::chrono::duration_cast<std::chrono::microseconds>(end-start).count());
    }
    // ##########################

    std::sort(runtimes.begin(), runtimes.end());
    
    std::fill(C, C+N*N, 0);
    hipMemcpyAsync(C, gpu_C2, (N * N) * sizeof(dtype), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    std::copy(C, C+N*N, result2);


    hipFree(gpu_A);
    hipFree(gpu_C1);
    hipFree(gpu_C2);
    hipFree(gpu_B);

    std::cout << "Done." << std::endl;
    std::cout << "Correctness Check" << std::endl;

    bool correct = true;
    dtype norm2_baseline = 0;
    dtype norm2_fused = 0;

    double tol = 1e-5;
    for(int i=0; i<N; ++i){
        for(int j=0; j<N; ++j){
            norm2_baseline += result1[i*N+j] * result1[i*N+j];
            norm2_fused += result2[i*N+j] * result2[i*N+j];
            if(std::abs(result1[i*N+j] - result2[i*N+j]) > tol){
                correct = false;
            }
        }
    }

    std::cout << "Evaluation     = "     << correct << std::endl;
    std::cout << "Norm2 Baseline = " << norm2_baseline << std::endl;
    std::cout << "Norm2 Fused    = "      << norm2_fused << std::endl;
}

int main(){
    run();
}
