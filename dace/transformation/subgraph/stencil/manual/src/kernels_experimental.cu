#include "hip/hip_runtime.h"
#include "kernels.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void kernel1(const dtype * __restrict__ gpu_A, dtype * __restrict__ B, int N) {
    {
        {
            int j = ((blockIdx.x * 32 + threadIdx.x) + 1);
            int i = ((blockIdx.y * 1 + threadIdx.y) + 1);
            if (j >= 1 && j < (N - 1)) {
                if (i >= 1) {
                    {
                        dtype a1 = gpu_A[((N * i) + j)];
                        dtype a2 = gpu_A[(((N * i) + j) - 1)];
                        dtype a3 = gpu_A[(((N * i) + j) + 1)];
                        dtype a4 = gpu_A[((N * (i + 1)) + j)];
                        dtype a5 = gpu_A[((N * (i - 1)) + j)];
                        dtype b;

                        ///////////////////
                        // Tasklet code (a)
                        b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
                        ///////////////////
                        B[((N * i) + j)] = b;
                    }
                }
            }
        }
    }
}


void run_kernel1(const dtype * __restrict__ gpu_A, dtype * __restrict__ gpu_B, int N, hipStream_t stream){

    dim3 grid_sz = dim3(int_ceil(int_ceil((N - 2), 1), 32), int_ceil(int_ceil((N - 2), 1), 1), int_ceil(1, 1));
    dim3 block_sz = dim3(32, 1, 1);
    kernel1<<<grid_sz, block_sz, 0, stream>>>(gpu_A, gpu_B, N);
}
__global__ void kernel2(const dtype * __restrict__ B, dtype * __restrict__ gpu_C, int N) {
    {
        {
            int j = ((blockIdx.x * 32 + threadIdx.x) + 2);
            int i = ((blockIdx.y * 1 + threadIdx.y) + 2);
            if (j >= 2 && j < (N - 2)) {
                if (i >= 2) {
                    {
                        dtype a1 = B[((N * i) + j)];
                        dtype a2 = B[(((N * i) + j) - 1)];
                        dtype a3 = B[(((N * i) + j) + 1)];
                        dtype a4 = B[((N * (i + 1)) + j)];
                        dtype a5 = B[((N * (i - 1)) + j)];
                        dtype b;

                        ///////////////////
                        // Tasklet code (b)
                        b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
                        ///////////////////
                        gpu_C[((N * i) + j)] = b;
                    }
                }
            }
        }
    }
}

void run_kernel2(const dtype * __restrict__ gpu_B, dtype * __restrict__ gpu_C, int N, hipStream_t stream)
{
    dim3 grid_sz = dim3(int_ceil(int_ceil((N - 4), 1), 32), int_ceil(int_ceil((N - 4), 1), 1), int_ceil(1, 1));
    dim3 block_sz = dim3(32, 1, 1);
    kernel2<<<grid_sz, block_sz, 0, stream>>>(gpu_B, gpu_C, N);

}
__device__ __forceinline__ void state1(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

__device__ __forceinline__ void state2(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

__device__ __forceinline__ void state3(const float * __in_gpu_A, const float * __in_gpu_A_0, const float * __in_gpu_A_1, const float * __in_gpu_A_2, const float * __in_gpu_A_3, float* __out_B, int N, int stencil_j) {

    {
        
        
        {
            float a1 = __in_gpu_A[0];
            float a2 = __in_gpu_A_0[0];
            float a3 = __in_gpu_A_1[2];
            float a4 = __in_gpu_A_2[(N + 1)];
            float a5 = __in_gpu_A_3[1];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[0] = b;
        }
    }
    {
        
        
        {
            float a1 = __in_gpu_A[1];
            float a2 = __in_gpu_A_0[1];
            float a3 = __in_gpu_A_1[3];
            float a4 = __in_gpu_A_2[(N + 2)];
            float a5 = __in_gpu_A_3[2];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[1] = b;

        }

    }
    {
        
        
        {
            float a1 = __in_gpu_A[2];
            float a2 = __in_gpu_A_0[2];
            float a3 = __in_gpu_A_1[4];
            float a4 = __in_gpu_A_2[(N + 3)];
            float a5 = __in_gpu_A_3[3];
            float b;

            ///////////////////
            // Tasklet code (a)
            b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
            ///////////////////

            __out_B[2] = b;
        }
    }
    
}

__device__ __forceinline__ void caller(const float * __in_gpu_A, float* __out_B, int N, int stencil_i, int stencil_j) {

    {
        
        
        state1(&__in_gpu_A[(N + 1)], &__in_gpu_A[N], &__in_gpu_A[N], &__in_gpu_A[N], &__in_gpu_A[0], &__out_B[0], N, stencil_j);
    }
    {
        
        
        state2(&__in_gpu_A[((2 * N) + 1)], &__in_gpu_A[(2 * N)], &__in_gpu_A[(2 * N)], &__in_gpu_A[(2 * N)], &__in_gpu_A[N], &__out_B[3], N, stencil_j);
    }
    {
        
        
        state3(&__in_gpu_A[((3 * N) + 1)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(3 * N)], &__in_gpu_A[(2 * N)], &__out_B[6], N, stencil_j);
    }
    
}



__global__ void outer_fused_0_0_9(const float * __restrict__ gpu_A, float * __restrict__ gpu_C, int N) {
    {
        {
            int stencil_j = (blockIdx.x * 32 + threadIdx.x);
            int stencil_i = (blockIdx.y * 1 + threadIdx.y);
            float *B = new float[9];
            memset(B, 0, sizeof(float)*9);
            if (stencil_j < (N - 4)) {
                {
                    //caller(&gpu_A[((N * stencil_i) + stencil_j)], &B[0], N, stencil_i, stencil_j);
                    //this is not a joke
                    {
                        for (auto i = (stencil_i + 2); i < (stencil_i + 3); i += 1) {
                            for (auto j = (stencil_j + 2); j < (stencil_j + 3); j += 1) {
                                {
                                    float a1 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 4)];
                                    float a2 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 5)];
                                    float a3 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 3)];
                                    float a4 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 1)];
                                    float a5 = B[(((((3 * i) + j) - (3 * stencil_i)) - stencil_j) - 7)];
                                    float b;

                                    ///////////////////
                                    // Tasklet code (b)
                                    b = (dtype(0.2) * ((((a1 + a2) + a3) + a4) + a5));
                                    ///////////////////

                                    gpu_C[((N * i) + j)] = b;
                                }
                            }
                        }
                    }
                }
            }
            delete[] B;
        }
    }
}

void run_fused(const float * __restrict__ gpu_A, float * __restrict__ gpu_C, int N, hipStream_t stream){
    outer_fused_0_0_9<<<dim3(int_ceil(int_ceil((N - 4), 1), 32), int_ceil(int_ceil((N - 4), 1), 1), int_ceil(1, 1)), dim3(32, 1, 1), 0, stream>>>(gpu_A, gpu_C, N);
    hipError_t error = hipGetLastError();
    if (error != 0){
        printf("ERROR in run_fused");
    }
}
